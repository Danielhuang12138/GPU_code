#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include <hipblas.h>

#include <iostream>
#include <stdio.h>
#define DATATYPE float
#define arraySize 5
#define threadnum 16
#define blocknum 8
#define arrayNsize 10
#define arrayMsize 15
#define single 1
hipError_t addWithCuda(int* c, int* a, int* b, int size);
//ʾ������
__global__ void addKernel(int *c, int *a, int *b)
{
    int i = threadIdx.x;
    c[i] = a[i] + b[i];
}

//��block��thread�����ӷ�
__global__ void vector_add_gpu_1(DATATYPE* a, DATATYPE* b, DATATYPE* c, int n) {
    for (int i = 0; i < n; ++i) {
        c[i] = a[i] + b[i];
    }
}

//��block��thread�����ӷ�
__global__ void vector_add_gpu_2(DATATYPE* a, DATATYPE* b, DATATYPE* c, int n) {
    int tid = threadIdx.x;
    const int t_n = blockDim.x;//�߳�����
    while (tid < n) {
        c[tid] = a[tid] + b[tid];
        tid += t_n;
    }
}

//��block��thread�����ӷ�
__global__ void vector_add_gpu_3(DATATYPE* a, DATATYPE* b, DATATYPE* c, int n) {
    //ȫ���߳�������tid=blockIdx.x*blockDim.x+threadIdx.x��������С��gird������thread����(gridDim.x*blockDim.x)
    const int tidx = threadIdx.x;
    const int bidx = blockIdx.x;
    const int t_n = gridDim.x * blockDim.x;
    int tid = bidx * blockDim.x + tidx;
    
    while (tid < n) {
        c[tid] = a[tid] + b[tid];
        tid += t_n;
    }
}
//m��n�����м���
void vector_add_mn(DATATYPE** a, DATATYPE** b, DATATYPE** c, int m, int n) {
    for (int i = 0; i < m; ++i) {
        for (int j = 0; j < n; ++j) {
            c[i][j] = a[i][j] + b[i][j];
        }
    }
}

//ά��Ϊm �� n�ľ���ӷ����м���
__global__ void vector_add_gpu_4(DATATYPE (* a)[arrayNsize], DATATYPE(* b)[arrayNsize], DATATYPE(* c)[arrayNsize], int m, int n) {
    unsigned int tid = threadIdx.x + blockDim.x * blockIdx.x;
    unsigned int tidy = threadIdx.y + blockDim.y * blockIdx.y;
    while (tid < m && tidy < m) {
        c[tid][tidy] = a[tid][tidy] + b[tid][tidy];
    }
}
//CPU���������ڻ�����
void vector_dot_product_serial(DATATYPE* a, DATATYPE* b, DATATYPE* c, int n) {
    double temp = 0;
    for (int i = 0; i < n; ++i) {
        temp += a[i] * b[i];
    }
    *c = temp;
}

//GPU��ɢ��Լ�����ڻ�
__global__ void vector_dot_product_gpu_1(DATATYPE* a, DATATYPE* b, DATATYPE* c, int n) {
    __shared__ DATATYPE tmp[threadnum];
    const int tidx = threadIdx.x;
    const int t_n = blockDim.x;
    int tid = tidx;
    double temp = 0.0;
    while (tid < n) {
        temp += a[tid] * b[tid];
        tid += t_n;
        tmp[tidx] = temp;
        __syncthreads();
        int i = 2, j = 1;
        while (i <= threadnum) {
            if ((tidx % i) == 0) {
                tmp[tidx] += tmp[tidx + j];
            }
            __syncthreads();
            i *= 2;
            j *= 2;
        }
        if (tidx == 0) {
            c[0] = tmp[0];
        }
    }

}

//��block���̹߳�Լ�����ڻ�
__global__ void vector_dot_product_gpu_2(DATATYPE* a, DATATYPE* b, DATATYPE* c, int n) {
    __shared__ DATATYPE tmp[threadnum];
    const int tidx = threadIdx.x;
    const int t_n = blockDim.x;
    int tid = tidx;
    double temp = 0.0;
    while (tid < n) {
        temp += a[tid] * b[tid];
        tid += t_n;
    }
    tmp[tidx] = temp;
    __syncthreads();
    int i = threadnum / 2;
    while (i != 0) {
        if (tidx < i) {
            tmp[tidx] += tmp[tidx + i];
        }
        __syncthreads();
        i /= 2;
    }
    if (tidx == 0) {
        c[0] = tmp[0];
    }
}

//��block�����ڻ���CPU���ι�Լ��
__global__ void vector_dot_product_gpu_3(DATATYPE* a, DATATYPE* b, DATATYPE* c_tmp, int n) {
    __shared__ DATATYPE tmp[threadnum];
    const int tidx = threadIdx.x;
    const int bidx = blockIdx.x;
    const int t_n = blockDim.x * gridDim.x;
    int tid = bidx * blockDim.x + tidx;
    double temp = 0.0;
    while (tid < n) {
        temp += a[tid] * b[tid];
        tid += t_n;
    }
    tmp[tidx] = temp;
    __syncthreads();
    int i = threadnum / 2;
    while (i != 0) {
        if (tidx < i) {
            tmp[tidx] += tmp[tidx + i];
        }
        __syncthreads();
        i /= 2;
    }
    if (tidx == 0) {
        c_tmp[bidx] = tmp[0];
    }
}

//GPU��Լ
__global__ void vector_dot_product_gpu_4 (float* result_tmp, float* result) {
    __shared__ float temp[blocknum];
    const int tidx = threadIdx.x;
    temp[tidx] = result_tmp[tidx];
    __syncthreads();
    int i = blocknum / 2;
    while (i != 0) {
        if (tidx < i) {
            temp[tidx] += temp[tidx + i];
        }
        __syncthreads();
        i /= 2;
    }
    if (tidx == 0) {
        result[0] = temp[0];
    }
}

//ԭ�Ӳ�����block�����ڻ�(���ι�Լ�滻һ��ԭ�Ӳ�����
__global__ void vector_dot_product_gpu_5_0(DATATYPE* a, DATATYPE* b, DATATYPE* c, int n) {
    if ((threadIdx.x == 0) && (blockIdx.x == 0)) {
        c[0] = 0.0;
    }
    const int tidx = threadIdx.x;
    const int bidx = blockIdx.x;
    const int t_n = blockDim.x * gridDim.x;
    int tid = bidx * blockDim.x + tidx;
    double temp = 0.0;
    while (tid < n) {
        temp += a[tid] * b[tid];
        tid += t_n;
    }
    atomicAdd(c, temp);
}
//ԭ�Ӳ�����block�����ڻ���block�ڹ�Լblock��ԭ�Ӳ�����
__global__ void vector_dot_product_gpu_5(DATATYPE* a, DATATYPE* b, DATATYPE* c, int n) {
    if ((threadIdx.x == 0) && (blockIdx.x == 0)) {
        c[0] = 0.0;
    }
    __shared__ DATATYPE tmp[threadnum];
    const int tidx = threadIdx.x;
    const int bidx = blockIdx.x;
    const int t_n = blockDim.x * gridDim.x;
    int tid = bidx * blockDim.x + tidx;
    double temp = 0.0;
    while (tid < n) {
        temp += a[tid] * b[tid];
        tid += t_n;
    }
    tmp[tidx] = temp;
    __syncthreads();
    int i = blockDim.x / 2;
    while (i != 0) {
        if (tidx < i) {
            tmp[tidx] += tmp[tidx + i];
        }
        __syncthreads();
        i /= 2;
    }
    if (tidx == 0) {
        atomicAdd(c, tmp[0]);
    }

}

//���������ӷ�
void vector_add_serial(DATATYPE* a, DATATYPE* b, DATATYPE* c, int n) {
    for (int i = 0; i < n; ++i) {
        c[i] = a[i] + b[i];
    }
}
int main()
{
    //int arraySize = 5;
    //dim3 blocknum(1);
    //dim3 threadnum(arrayMsize,arrayNsize);
    float a[arraySize] = { 1, 2, 3, 4, 5 };
    float b[arraySize] = { 10, 20, 30, 40, 50 };
    float c[arraySize] = { 0 };
    float c1[arraySize] = { 0 };
    float **aa;
    float **bb;
    float **cc;
    aa = (float**)malloc(sizeof(float*) * arrayMsize);
    bb = (float**)malloc(sizeof(float*) * arrayMsize);
    cc = (float**)malloc(sizeof(float*) * arrayMsize);
    for (int i = 0; i < arrayMsize; ++i) {
        aa[i] = (float*)malloc(sizeof(float*) * arrayNsize);
        bb[i] = (float*)malloc(sizeof(float*) * arrayNsize);
        cc[i] = (float*)malloc(sizeof(float*) * arrayNsize);

    }
    for (int i = 0; i < arrayMsize; ++i) {
        for (int j = 0; j < arrayNsize; ++j) {
            aa[i][j] = j;
            bb[i][j] = j * 10;
            cc[i][j] = 0;
        }
    }
    //���в���
    vector_add_serial(a, b, c, arraySize);
    //printf("serial :{1,2,3,4,5} + {10,20,30,40,50} = {%f,%f,%f,%f,%f}\n",c[0], c[1], c[2], c[3], c[4]);
    for (int i = 0; i < arraySize; ++i) {
        c[i] = 0;
    }

    //��block��thread�ӷ�����
        //GPU�ڴ����
    DATATYPE* d_a, * d_b, * d_c;
    hipMalloc((void**)&d_a, sizeof(DATATYPE) * arraySize);
    hipMalloc((void**)&d_b, sizeof(DATATYPE) * arraySize);
    hipMalloc((void**)&d_c, sizeof(DATATYPE) * arraySize);
        //�������ݵ�GPU
    hipMemcpy(d_a, a, sizeof(DATATYPE) * arraySize, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, sizeof(DATATYPE) * arraySize, hipMemcpyHostToDevice);
        //����
    vector_add_gpu_1<<<single,single>>>(d_a, d_b, d_c, arraySize);
        //���ƽ����CPU
    hipMemcpy(c, d_c, sizeof(DATATYPE) * arraySize, hipMemcpyDeviceToHost);
        //�ͷſռ�
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    //printf("single block single thread :{1,2,3,4,5} + {10,20,30,40,50} = {%f,%f,%f,%f,%f}\n",c[0], c[1], c[2], c[3], c[4]);
    for (int i = 0; i < arraySize; ++i) {
        c[i] = 0;
    }

    //��block��thread�ӷ�
    hipMalloc((void**)&d_a, sizeof(DATATYPE) * arraySize);
    hipMalloc((void**)&d_b, sizeof(DATATYPE) * arraySize);
    hipMalloc((void**)&d_c, sizeof(DATATYPE) * arraySize);
        //�������ݵ�GPU
    hipMemcpy(d_a, a, sizeof(DATATYPE) * arraySize, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, sizeof(DATATYPE) * arraySize, hipMemcpyHostToDevice);
        //����
    vector_add_gpu_2 <<<1, threadnum >>> (d_a, d_b, d_c, arraySize);
        //���ƽ����CPU
    hipMemcpy(c, d_c, sizeof(DATATYPE) * arraySize, hipMemcpyDeviceToHost);
        //�ͷſռ�
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    //printf("single block multiple thread :{1,2,3,4,5} + {10,20,30,40,50} = {%f,%f,%f,%f,%f}\n",c[0], c[1], c[2], c[3], c[4]);
    for (int i = 0; i < arraySize; ++i) {
        c[i] = 0;
    }

    //��block��thread�ӷ�
    hipMalloc((void**)&d_a, sizeof(DATATYPE) * arraySize);
    hipMalloc((void**)&d_b, sizeof(DATATYPE) * arraySize);
    hipMalloc((void**)&d_c, sizeof(DATATYPE) * arraySize);
        //�������ݵ�GPU
    hipMemcpy(d_a, a, sizeof(DATATYPE) * arraySize, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, sizeof(DATATYPE) * arraySize, hipMemcpyHostToDevice);
        //����
    vector_add_gpu_3 <<<blocknum, threadnum >>> (d_a, d_b, d_c, arraySize);
        //���ƽ����CPU
    hipMemcpy(c, d_c, sizeof(DATATYPE) * arraySize, hipMemcpyDeviceToHost);
        //�ͷſռ�
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    //printf("multiple block multiple thread :{1,2,3,4,5} + {10,20,30,40,50} = {%f,%f,%f,%f,%f}\n",c[0], c[1], c[2], c[3], c[4]);
    for (int i = 0; i < arraySize; ++i) {
        c[i] = 0;
    }
    //cublas�������ӷ�
    DATATYPE* d_aa, * d_bb;
    hipblasHandle_t handle;
    hipblasCreate(&handle);
    hipMalloc((void**)&d_aa, sizeof(DATATYPE) * arraySize);
    hipMalloc((void**)&d_bb, sizeof(DATATYPE) * arraySize);
    float alpha = 1.0;
    hipblasSetVector(arraySize, sizeof(DATATYPE), a, 1, d_aa, 1);
    hipblasSetVector(arraySize, sizeof(DATATYPE), b, 1, d_bb, 1);
    hipblasSaxpy(handle, arraySize, &alpha, d_aa, 1, d_bb, 1);
    hipblasGetVector(arraySize, sizeof(DATATYPE), d_bb, 1, c1, 1);
    hipFree(d_aa);
    hipFree(d_bb);
    hipblasDestroy(handle);
    //printf("cublas :{1,2,3,4,5} + {10,20,30,40,50} = {%f,%f,%f,%f,%f}\n",c1[0], c1[1], c1[2], c1[3], c1[4]);
    for (int i = 0; i < arraySize; ++i) {
        c1[i] = 0;
    }

    //m��n�����мӷ�
    /*DATATYPE(*d_aaa)[arrayNsize], (*d_bbb)[arrayNsize], (*d_ccc)[arrayNsize];
    hipMalloc((void**)&d_aaa, sizeof(DATATYPE) * arrayMsize * arrayNsize);
    hipMalloc((void**)&d_bbb, sizeof(DATATYPE) * arrayMsize * arrayNsize);
    hipMalloc((void**)&d_ccc, sizeof(DATATYPE) * arrayMsize * arrayNsize);
    hipMemcpy(d_aaa, aa, sizeof(DATATYPE) * arrayNsize * arrayMsize, hipMemcpyHostToDevice);
    hipMemcpy(d_bbb, bb, sizeof(DATATYPE)* arrayNsize* arrayMsize, hipMemcpyHostToDevice);
    hipMemcpy(d_ccc, cc, sizeof(DATATYPE)* arrayNsize* arrayMsize, hipMemcpyHostToDevice);
    vector_add_gpu_4 << <blocknum, threadnum >> > (d_aaa, d_bbb, d_ccc, arrayMsize, arrayNsize);
    hipMemcpy(cc, d_ccc, sizeof(DATATYPE)* arrayNsize*arrayMsize, hipMemcpyDeviceToHost);
    hipFree(d_aaa);
    hipFree(d_bbb);
    hipFree(d_ccc);
    std::cout << "m��n matrix add\n";
    for (int i = 0; i < arrayMsize; ++i) {
        for (int j = 0; j < arrayNsize; ++j) {
            std::cout << cc[i][j] << " ";
            cc[i][j] = 0;
        }
        std::cout << "\n";
    }
    */
    //m��n�ӷ������֤
    vector_add_mn(aa, bb, cc, arrayMsize, arrayNsize);
    //std::cout << "m��n matrix valid\n";
    /*for (int i = 0; i < arrayMsize; ++i) {
        for (int j = 0; j < arrayNsize; ++j) {
            std::cout << cc[i][j] << " ";
            cc[i][j] = 0;
        }
        std::cout << "\n";
    }*/

    //��block��ɢ��Լ�����ڻ�
    DATATYPE* d_cccc,*d_ca;
    DATATYPE ccccd,*cccc;
    cccc = &ccccd;
    hipMalloc((void**)&d_a, sizeof(DATATYPE) * arraySize);
    hipMalloc((void**)&d_b, sizeof(DATATYPE) * arraySize);
    hipMalloc((void**)&d_cccc, sizeof(DATATYPE));
    //�������ݵ�GPU
    hipMemcpy(d_a, a, sizeof(DATATYPE) * arraySize, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, sizeof(DATATYPE) * arraySize, hipMemcpyHostToDevice);
    //����
    vector_dot_product_gpu_1 << <single, threadnum >> > (d_a, d_b, d_c, arraySize);
    //���ƽ����CPU
    hipMemcpy(cccc, d_c, sizeof(DATATYPE), hipMemcpyDeviceToHost);
    //�ͷſռ�
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    printf("single block:{1,2,3,4,5} �� {10,20,30,40,50} = {%f}\n",ccccd);
    cccc = 0;

    //��block���̹߳�Լ�����ڻ�
    hipMalloc((void**)&d_a, sizeof(DATATYPE) * arraySize);
    hipMalloc((void**)&d_b, sizeof(DATATYPE) * arraySize);
    hipMalloc((void**)&d_c, sizeof(DATATYPE) * arraySize);
    //�������ݵ�GPU
    hipMemcpy(d_a, a, sizeof(DATATYPE) * arraySize, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, sizeof(DATATYPE) * arraySize, hipMemcpyHostToDevice);
    //����
    vector_dot_product_gpu_2 << <single, threadnum >> > (d_a, d_b, d_c, arraySize);
    //���ƽ����CPU
    hipMemcpy(c, d_c, sizeof(DATATYPE) * arraySize, hipMemcpyDeviceToHost);
    //�ͷſռ�
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    printf("single block:{1,2,3,4,5} �� {10,20,30,40,50} = {%f}\n",c[0]);
    c[0] = 0;

    //��block�����ڻ���CPU���ι�Լ��
    hipMalloc((void**)&d_a, sizeof(DATATYPE) * arraySize);
    hipMalloc((void**)&d_b, sizeof(DATATYPE) * arraySize);
    hipMalloc((void**)&d_c, sizeof(DATATYPE) * arraySize);
    //�������ݵ�GPU
    hipMemcpy(d_a, a, sizeof(DATATYPE) * arraySize, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, sizeof(DATATYPE) * arraySize, hipMemcpyHostToDevice);
    //����
    vector_dot_product_gpu_3 << <blocknum, threadnum >> > (d_a, d_b, d_c, arraySize);
    //���ƽ����CPU
    hipMemcpy(c, d_c, sizeof(DATATYPE) * arraySize, hipMemcpyDeviceToHost);
    //�ͷſռ�
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    double temp=0;
    for (int i = 0; i < blocknum&&i<arraySize; i++) {
        if(c[i]!=NULL)
        temp += c[i];
    }
    c[0] = temp;
    printf("multiple block(CPU):{1,2,3,4,5} �� {10,20,30,40,50} = {%f}\n", c[0]);
    c[0] = 0;
    for (int i = 0; i < arraySize; ++i) {
        c[i] = 0;
    }
    //��block�����ڻ���GPU���ι�Լ��
    hipMalloc((void**)&d_a, sizeof(DATATYPE) * arraySize);
    hipMalloc((void**)&d_b, sizeof(DATATYPE) * arraySize);
    hipMalloc((void**)&d_c, sizeof(DATATYPE) * arraySize);
    hipMalloc((void**)&d_ca, sizeof(DATATYPE) * arraySize);
    //�������ݵ�GPU
    hipMemcpy(d_a, a, sizeof(DATATYPE) * arraySize, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, sizeof(DATATYPE) * arraySize, hipMemcpyHostToDevice);
    //����
    vector_dot_product_gpu_3 << <blocknum, threadnum >> > (d_a, d_b, d_c, arraySize);
    vector_dot_product_gpu_4 <<< 1, blocknum >> > (d_c, d_ca);
    //���ƽ����CPU
    hipMemcpy(c, d_ca, sizeof(DATATYPE) * arraySize, hipMemcpyDeviceToHost);
    //�ͷſռ�
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    hipFree(d_ca);
    printf("multiple block(GPU):{1,2,3,4,5} �� {10,20,30,40,50} = {%f}\n", c[0]);
    c[0] = 0;
    for (int i = 0; i < arraySize; ++i) {
        c[i] = 0;
    }

    //ԭ�Ӳ���
    hipMalloc((void**)&d_a, sizeof(DATATYPE) * arraySize);
    hipMalloc((void**)&d_b, sizeof(DATATYPE) * arraySize);
    hipMalloc((void**)&d_c, sizeof(DATATYPE) * arraySize);
    //�������ݵ�GPU
    hipMemcpy(d_a, a, sizeof(DATATYPE) * arraySize, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, sizeof(DATATYPE) * arraySize, hipMemcpyHostToDevice);
    //����
    vector_dot_product_gpu_5_0 << <blocknum, threadnum >> > (d_a, d_b, d_c, arraySize);
    //���ƽ����CPU
    hipMemcpy(c, d_c, sizeof(DATATYPE) * arraySize, hipMemcpyDeviceToHost);
    //�ͷſռ�
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    printf("atomic_0 multiple block(GPU):{1,2,3,4,5} �� {10,20,30,40,50} = {%f}\n", c[0]);
    c[0] = 0;
    for (int i = 0; i < arraySize; ++i) {
        c[i] = 0;
    }

    hipMalloc((void**)&d_a, sizeof(DATATYPE) * arraySize);
    hipMalloc((void**)&d_b, sizeof(DATATYPE) * arraySize);
    hipMalloc((void**)&d_c, sizeof(DATATYPE) * arraySize);
    //�������ݵ�GPU
    hipMemcpy(d_a, a, sizeof(DATATYPE) * arraySize, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, sizeof(DATATYPE) * arraySize, hipMemcpyHostToDevice);
    //����
    vector_dot_product_gpu_5 << <blocknum, threadnum >> > (d_a, d_b, d_c, arraySize);
    //���ƽ����CPU
    hipMemcpy(c, d_c, sizeof(DATATYPE) * arraySize, hipMemcpyDeviceToHost);
    //�ͷſռ�
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    printf("atomic_1 multiple block(GPU):{1,2,3,4,5} �� {10,20,30,40,50} = {%f}\n", c[0]);
    c[0] = 0;
    for (int i = 0; i < arraySize; ++i) {
        c[i] = 0;
    }


    // Add vectors in parallel.
    /*hipError_t cudaStatus = addWithCuda(c, a, b, arraySize);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addWithCuda failed!");
        return 1;
    }

    printf("{1,2,3,4,5} + {10,20,30,40,50} = {%f,%f,%f,%f,%f}\n",
        c[0], c[1], c[2], c[3], c[4]);

    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }
    */
    return 0;
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t addWithCuda(int *c, int *a, int *b, int size)
{
    int *dev_a = 0;
    int *dev_b = 0;
    int *dev_c = 0;
    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    // Launch a kernel on the GPU with one thread for each element.
    addKernel<<<1, size>>>(dev_c, dev_a, dev_b);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    
    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
    hipFree(dev_c);
    hipFree(dev_a);
    hipFree(dev_b);
    
    return cudaStatus;
}
